#include "hip/hip_runtime.h"
#include <hip/driver_types.h>
#include "../../../common/common.h"
#include "../../../app/arguments.h"

#include "../../hasher.h"
#include "../../argon2/argon2.h"

#include "cuda_hasher.h"

#define ITEMS_PER_SEGMENT               32
#define BLOCK_SIZE_ULONG                128
#define KERNEL_WORKGROUP_SIZE   		32

__device__ uint64_t upsample(uint32_t hi, uint32_t lo)
{
	return ((uint64_t)hi << 32) | (uint64_t)lo;
}

__device__ uint64_t rotate(uint64_t x, uint32_t n)
{
	return (x >> (64-n)) | (x << n);
}

#define fBlaMka(x, y) ((x) + (y) + 2 * upsample(__umulhi((uint32_t)(x), (uint32_t)(y)), (uint32_t)(x) * (uint32_t)y))

#if defined(USE_NON_PTX_CUDA)

	#define COMPUTE            \
		a = fBlaMka(a, b);          \
		d = rotate(d ^ a, 32);      \
		c = fBlaMka(c, d);          \
		b = rotate(b ^ c, 40);      \
		a = fBlaMka(a, b);          \
		d = rotate(d ^ a, 48);      \
		c = fBlaMka(c, d);          \
		b = rotate(b ^ c, 1);
#else

    #define COMPUTE            \
	asm ("{"  \
	     ".reg .u64 d1, d2, a, b, c, d;\n\t"     \
		 ".reg .u32 s1, s2, s3, s4;\n\t"     \
		 "add.u64 d1, %0, %1;\n\t"     \
		 "cvt.u32.u64 s1, %0;\n\t"     \
		 "cvt.u32.u64 s2, %1;\n\t"     \
		 "mul.lo.u32 s3, s1, s2;\n\t"     \
		 "mul.hi.u32 s4, s1, s2;\n\t"     \
		 "mov.b64 a, {s3, s4};\n\t"     \
		 "shl.b64 d2, a, 1;\n\t"     \
		 "add.u64 a, d1, d2;\n\t"     \
		 "xor.b64 d1, %3, a;\n\t"     \
		 "mov.b64 {s1, s2}, d1;\n\t"     \
		 "mov.b64 d, {s2, s1};\n\t"     \
		 "add.u64 d1, %2, d;\n\t"     \
		 "cvt.u32.u64 s1, %2;\n\t"     \
		 "mul.lo.u32 s3, s1, s2;\n\t"     \
		 "mul.hi.u32 s4, s1, s2;\n\t"     \
		 "mov.b64 c, {s3, s4};\n\t"     \
		 "shl.b64 d2, c, 1;\n\t"     \
		 "add.u64 c, d1, d2;\n\t"     \
		 "xor.b64 d1, %1, c;\n\t"     \
		 "mov.b64 {s3, s4}, d1;\n\t"     \
		 "prmt.b32 s2, s3, s4, 0x6543;\n\t"     \
		 "prmt.b32 s1, s3, s4, 0x2107;\n\t"     \
		 "mov.b64 b, {s2, s1};\n\t"     \
		 "add.u64 d1, a, b;\n\t"     \
		 "cvt.u32.u64 s1, a;\n\t"     \
		 "mul.lo.u32 s3, s1, s2;\n\t"     \
		 "mul.hi.u32 s4, s1, s2;\n\t"     \
		 "mov.b64 a, {s3, s4};\n\t"     \
		 "shl.b64 d2, a, 1;\n\t"     \
		 "add.u64 %0, d1, d2;\n\t"     \
		 "xor.b64 d1, d, %0;\n\t"     \
		 "mov.b64 {s3, s4}, d1;\n\t"     \
		 "prmt.b32 s2, s3, s4, 0x5432;\n\t"     \
		 "prmt.b32 s1, s3, s4, 0x1076;\n\t"     \
		 "mov.b64 %3, {s2, s1};\n\t"     \
		 "add.u64 d1, c, %3;\n\t"     \
		 "cvt.u32.u64 s1, c;\n\t"     \
		 "mul.lo.u32 s3, s1, s2;\n\t"     \
		 "mul.hi.u32 s4, s1, s2;\n\t"     \
		 "mov.b64 c, {s3, s4};\n\t"     \
		 "shl.b64 d2, c, 1;\n\t"     \
		 "add.u64 %2, d1, d2;\n\t"     \
		 "xor.b64 d1, b, %2;\n\t"     \
		 "shl.b64 a, d1, 1;\n\t"     \
		 "shr.b64 b, d1, 63;\n\t"     \
		 "add.u64 %1, a, b;\n\t" \
	"}" : "+l"(a), "+l"(b), "+l"(c), "+l"(d));

#endif

#define G1(data)           \
{                           \
	COMPUTE \
	data[i1_1] = b; \
    data[i1_2] = c; \
    data[i1_3] = d; \
    __syncthreads(); \
}

#define G2(data)           \
{ \
    b = data[i2_1]; \
    c = data[i2_2]; \
    d = data[i2_3]; \
	COMPUTE \
    data[i2_0] = a; \
    data[i2_1] = b; \
    data[i2_2] = c; \
    data[i2_3] = d; \
    __syncthreads(); \
}

#define G3(data)           \
{                           \
    a = data[i3_0]; \
    b = data[i3_1]; \
    c = data[i3_2]; \
    d = data[i3_3]; \
	COMPUTE \
	data[i3_1] = b; \
    data[i3_2] = c; \
    data[i3_3] = d; \
    __syncthreads(); \
}

#define G4(data)           \
{                           \
    b = data[i4_1]; \
    c = data[i4_2]; \
    d = data[i4_3]; \
	COMPUTE \
    data[i4_0] = a; \
    data[i4_1] = b; \
    data[i4_2] = c; \
    data[i4_3] = d; \
    __syncthreads(); \
    a = data[i1_0]; \
    b = data[i1_1]; \
    c = data[i1_2]; \
    d = data[i1_3]; \
}

#define copy_block(D, S) \
	dst = (uint32_t*)(D); \
	src = (uint32_t*)(S); \
	for(int k=0;k<8;k++)  \
		dst[id + 32 * k] = src[id + 32 * k];

int offsets[512] = {
		0, 4, 8, 12,
		1, 5, 9, 13,
		2, 6, 10, 14,
		3, 7, 11, 15,
		16, 20, 24, 28,
		17, 21, 25, 29,
		18, 22, 26, 30,
		19, 23, 27, 31,
		32, 36, 40, 44,
		33, 37, 41, 45,
		34, 38, 42, 46,
		35, 39, 43, 47,
		48, 52, 56, 60,
		49, 53, 57, 61,
		50, 54, 58, 62,
		51, 55, 59, 63,
		64, 68, 72, 76,
		65, 69, 73, 77,
		66, 70, 74, 78,
		67, 71, 75, 79,
		80, 84, 88, 92,
		81, 85, 89, 93,
		82, 86, 90, 94,
		83, 87, 91, 95,
		96, 100, 104, 108,
		97, 101, 105, 109,
		98, 102, 106, 110,
		99, 103, 107, 111,
		112, 116, 120, 124,
		113, 117, 121, 125,
		114, 118, 122, 126,
		115, 119, 123, 127,
		0, 5, 10, 15,
		1, 6, 11, 12,
		2, 7, 8, 13,
		3, 4, 9, 14,
		16, 21, 26, 31,
		17, 22, 27, 28,
		18, 23, 24, 29,
		19, 20, 25, 30,
		32, 37, 42, 47,
		33, 38, 43, 44,
		34, 39, 40, 45,
		35, 36, 41, 46,
		48, 53, 58, 63,
		49, 54, 59, 60,
		50, 55, 56, 61,
		51, 52, 57, 62,
		64, 69, 74, 79,
		65, 70, 75, 76,
		66, 71, 72, 77,
		67, 68, 73, 78,
		80, 85, 90, 95,
		81, 86, 91, 92,
		82, 87, 88, 93,
		83, 84, 89, 94,
		96, 101, 106, 111,
		97, 102, 107, 108,
		98, 103, 104, 109,
		99, 100, 105, 110,
		112, 117, 122, 127,
		113, 118, 123, 124,
		114, 119, 120, 125,
		115, 116, 121, 126,
		0, 32, 64, 96,
		1, 33, 65, 97,
		2, 34, 66, 98,
		3, 35, 67, 99,
		4, 36, 68, 100,
		5, 37, 69, 101,
		6, 38, 70, 102,
		7, 39, 71, 103,
		8, 40, 72, 104,
		9, 41, 73, 105,
		10, 42, 74, 106,
		11, 43, 75, 107,
		12, 44, 76, 108,
		13, 45, 77, 109,
		14, 46, 78, 110,
		15, 47, 79, 111,
		16, 48, 80, 112,
		17, 49, 81, 113,
		18, 50, 82, 114,
		19, 51, 83, 115,
		20, 52, 84, 116,
		21, 53, 85, 117,
		22, 54, 86, 118,
		23, 55, 87, 119,
		24, 56, 88, 120,
		25, 57, 89, 121,
		26, 58, 90, 122,
		27, 59, 91, 123,
		28, 60, 92, 124,
		29, 61, 93, 125,
		30, 62, 94, 126,
		31, 63, 95, 127,
		0, 33, 80, 113,
		1, 48, 81, 96,
		2, 35, 82, 115,
		3, 50, 83, 98,
		4, 37, 84, 117,
		5, 52, 85, 100,
		6, 39, 86, 119,
		7, 54, 87, 102,
		8, 41, 88, 121,
		9, 56, 89, 104,
		10, 43, 90, 123,
		11, 58, 91, 106,
		12, 45, 92, 125,
		13, 60, 93, 108,
		14, 47, 94, 127,
		15, 62, 95, 110,
		16, 49, 64, 97,
		17, 32, 65, 112,
		18, 51, 66, 99,
		19, 34, 67, 114,
		20, 53, 68, 101,
		21, 36, 69, 116,
		22, 55, 70, 103,
		23, 38, 71, 118,
		24, 57, 72, 105,
		25, 40, 73, 120,
		26, 59, 74, 107,
		27, 42, 75, 122,
		28, 61, 76, 109,
		29, 44, 77, 124,
		30, 63, 78, 111,
		31, 46, 79, 126
};

__global__ void fill_blocks_cpu(uint64_t *scratchpad,
							uint64_t *seed,
							uint64_t *out,
							uint64_t *addresses,
							int *offsets_,
							int memsize) {
	__shared__ uint64_t state[BLOCK_SIZE_ULONG];
	uint32_t *src, *dst;

	uint64_t a, b, c, d, x, y, z, w;

	int hash = blockIdx.x;
	int id = threadIdx.x;

	int offset = id << 2;

	int i1_0 = offsets_[offset];
	int i1_1 = offsets_[offset + 1];
	int i1_2 = offsets_[offset + 2];
	int i1_3 = offsets_[offset + 3];

	int i2_0 = offsets_[offset + 128];
	int i2_1 = offsets_[offset + 129];
	int i2_2 = offsets_[offset + 130];
	int i2_3 = offsets_[offset + 131];

	int i3_0 = offsets_[offset + 256];
	int i3_1 = offsets_[offset + 257];
	int i3_2 = offsets_[offset + 258];
	int i3_3 = offsets_[offset + 259];

	int i4_0 = offsets_[offset + 384];
	int i4_1 = offsets_[offset + 385];
	int i4_2 = offsets_[offset + 386];
	int i4_3 = offsets_[offset + 387];

	uint64_t *memory = scratchpad + hash * (memsize >> 3);

	uint64_t *out_mem = out + hash * 2 * BLOCK_SIZE_ULONG;
	uint64_t *mem_seed = seed + hash * 2 * BLOCK_SIZE_ULONG;

	uint64_t *seed_dst = memory;
	copy_block(seed_dst, mem_seed);
	mem_seed += BLOCK_SIZE_ULONG;
	seed_dst += BLOCK_SIZE_ULONG;
	copy_block(seed_dst, mem_seed);

	uint64_t *next_block;
	uint64_t *ref_block;

	uint64_t *stop_addr = addresses + 524286;

	a = seed_dst[i1_0];
	b = seed_dst[i1_1];
	c = seed_dst[i1_2];
	d = seed_dst[i1_3];

	for(; addresses < stop_addr; addresses += 32) {
		int64_t store_addr = addresses[id];
		uint64_t i_limit = stop_addr - addresses;
		if(i_limit > 32) i_limit = 32;

		for(int i=0;i<i_limit;i++) {
			uint64_t addr = __shfl_sync(0xFFFFFFFF, store_addr, i, 32);
			int32_t addr0, addr1;
			asm("mov.b64 {%0, %1}, %2;": "=r"(addr0), "=r"(addr1) : "l"(addr));

			if (addr0 != -1) {
				next_block = memory + addr0 * BLOCK_SIZE_ULONG;
			}
			ref_block = memory + addr1 * BLOCK_SIZE_ULONG;

			a ^= ref_block[i1_0];
			b ^= ref_block[i1_1];
			c ^= ref_block[i1_2];
			d ^= ref_block[i1_3];

			x = a; y = b; z = c; w = d;

			G1(state);
			G2(state);
			G3(state);
			G4(state);

			a ^= x; b ^= y; c ^= z; d ^= w;

			if (addr0 != -1) {
				next_block[i1_0] = a;
				next_block[i1_1] = b;
				next_block[i1_2] = c;
				next_block[i1_3] = d;
			}
		}
	}

	out_mem[i1_0] = a;
	out_mem[i1_1] = b;
	out_mem[i1_2] = c;
	out_mem[i1_3] = d;
};

__global__ void fill_blocks_gpu(uint64_t *scratchpad,
							uint64_t *seed,
							uint64_t *out,
							uint32_t *addresses,
							uint32_t *segments,
							int *offsets_,
							int memsize) {
	__shared__ uint64_t state[4 * BLOCK_SIZE_ULONG];

	uint64_t a, b, c, d, x, y, z, w;
	uint32_t *src, *dst;

	int hash = blockIdx.x;
	int local_id = threadIdx.x;

	int id = local_id % ITEMS_PER_SEGMENT;
	int segment = local_id / ITEMS_PER_SEGMENT;

	int offset = id << 2;

	int i1_0 = offsets_[offset];
	int i1_1 = offsets_[offset + 1];
	int i1_2 = offsets_[offset + 2];
	int i1_3 = offsets_[offset + 3];

	int i2_0 = offsets_[offset + 128];
	int i2_1 = offsets_[offset + 129];
	int i2_2 = offsets_[offset + 130];
	int i2_3 = offsets_[offset + 131];

	int i3_0 = offsets_[offset + 256];
	int i3_1 = offsets_[offset + 257];
	int i3_2 = offsets_[offset + 258];
	int i3_3 = offsets_[offset + 259];

	int i4_0 = offsets_[offset + 384];
	int i4_1 = offsets_[offset + 385];
	int i4_2 = offsets_[offset + 386];
	int i4_3 = offsets_[offset + 387];

	uint64_t *memory = scratchpad + hash * (memsize >> 3);

	uint64_t *out_mem = out + hash * 8 * BLOCK_SIZE_ULONG;
	uint64_t *mem_seed = seed + hash * 8 * BLOCK_SIZE_ULONG;

	uint64_t *seed_src = mem_seed + segment * 2 * BLOCK_SIZE_ULONG;
	uint64_t *seed_dst = memory + segment * 4096 * BLOCK_SIZE_ULONG;
	copy_block(seed_dst, seed_src);
	seed_src += BLOCK_SIZE_ULONG;
	seed_dst += BLOCK_SIZE_ULONG;
	copy_block(seed_dst, seed_src);

	uint64_t *next_block;
	uint64_t *prev_block;
	uint64_t *ref_block;

	uint64_t *local_state = state + segment * BLOCK_SIZE_ULONG;

	segments += segment;
	uint16_t addr_start_idx = 0;
	uint16_t prev_blk_idx;
	int inc = 1022;

	//without xor
	for(int s=0; s<4; s++) {
		uint32_t curr_seg = segments[s * 4];

		asm("mov.b32 {%0, %1}, %2;"
		: "=h"(addr_start_idx), "=h"(prev_blk_idx) : "r"(curr_seg));

		uint32_t *addr = addresses + addr_start_idx;
		uint32_t *stop_addr = addresses + addr_start_idx + inc;
		inc = 1024;

		prev_block = memory + prev_blk_idx * BLOCK_SIZE_ULONG;
		__syncthreads();

		a = prev_block[i1_0];
		b = prev_block[i1_1];
		c = prev_block[i1_2];
		d = prev_block[i1_3];

		for(; addr < stop_addr; addr += 32) {
			uint32_t store_addr = addr[id];

			uint64_t i_limit = stop_addr - addr;
			if(i_limit > 32) i_limit = 32;

			for(int i=0;i<i_limit;i++) {
				uint32_t local_addr = __shfl_sync(0xFFFFFFFF, store_addr, i, 32);
				int16_t addr0, addr1;
				asm("{mov.b32 {%0, %1}, %2;}": "=h"(addr0), "=h"(addr1) : "r"(local_addr));

				next_block = memory + addr0 * BLOCK_SIZE_ULONG;
				ref_block = memory + addr1 * BLOCK_SIZE_ULONG;

				copy_block(local_state, ref_block);

				a ^= local_state[i1_0];
				b ^= local_state[i1_1];
				c ^= local_state[i1_2];
				d ^= local_state[i1_3];

				x = a; y = b; z = c; w = d;

				G1(local_state);
				G2(local_state);
				G3(local_state);
				G4(local_state);

				a ^= x; b ^= y; c ^= z; d ^= w;

				next_block[i1_0] = a;
				next_block[i1_1] = b;
				next_block[i1_2] = c;
				next_block[i1_3] = d;
			}
		}
	}

	// with xor
	for(int s=4; s<16; s++) {
		uint32_t curr_seg = segments[s * 4];

		asm("mov.b32 {%0, %1}, %2;"
		: "=h"(addr_start_idx), "=h"(prev_blk_idx) : "r"(curr_seg));

		uint32_t *addr = addresses + addr_start_idx;
		uint32_t *stop_addr = addresses + addr_start_idx + 1024;

		prev_block = memory + prev_blk_idx * BLOCK_SIZE_ULONG;
		__syncthreads();

		a = prev_block[i1_0];
		b = prev_block[i1_1];
		c = prev_block[i1_2];
		d = prev_block[i1_3];

		for(; addr < stop_addr; addr += 32) {
			uint32_t store_addr = addr[id];

			for (int i = 0; i < 32; i++) {
				uint32_t local_addr = __shfl_sync(0xFFFFFFFF, store_addr, i, 32);
				int16_t addr0, addr1;
				asm("{mov.b32 {%0, %1}, %2;}": "=h"(addr0), "=h"(addr1) : "r"(local_addr));

				next_block = memory + addr0 * BLOCK_SIZE_ULONG;
				ref_block = memory + addr1 * BLOCK_SIZE_ULONG;

				copy_block(local_state, ref_block);

				a ^= local_state[i1_0];
				b ^= local_state[i1_1];
				c ^= local_state[i1_2];
				d ^= local_state[i1_3];

				x = a; y = b; z = c; w = d;

				x ^= next_block[i1_0];
				y ^= next_block[i1_1];
				z ^= next_block[i1_2];
				w ^= next_block[i1_3];

				G1(local_state);
				G2(local_state);
				G3(local_state);
				G4(local_state);

				a ^= x; b ^= y; c ^= z; d ^= w;

				next_block[i1_0] = a;
				next_block[i1_1] = b;
				next_block[i1_2] = c;
				next_block[i1_3] = d;
			}
		}
	}

	__syncthreads();

	int dst_addr = 65528;

	next_block = memory + ((int16_t*)(&addresses[dst_addr]))[0] * BLOCK_SIZE_ULONG;
	out_mem[local_id] = next_block[local_id];

	for(;dst_addr < 65531; ++dst_addr) {
		next_block = memory + ((int16_t*)(&addresses[dst_addr]))[1] * BLOCK_SIZE_ULONG;
		out_mem[local_id] ^= next_block[local_id];
	}
};

void cuda_allocate(cuda_device_info *device) {
	int max_threads = max(device->threads_profile_1_1_524288, device->threads_profile_4_4_16384);

	device->error = hipSetDevice(device->device_index);
	if(device->error != hipSuccess) {
		device->error_message = "Error setting current device for memory allocation.";
		return;
	}

	//optimise address sizes
	uint32_t *addresses_1_1_524288 = (uint32_t *)malloc(argon2profile_1_1_524288.block_refs_size * 2 * sizeof(int32_t));
	for(int i=0;i<argon2profile_1_1_524288.block_refs_size;i++) {
		addresses_1_1_524288[i*2] = argon2profile_1_1_524288.block_refs[i*3];
		addresses_1_1_524288[i*2+1] = argon2profile_1_1_524288.block_refs[i*3 + 2];
	}
	device->error = hipMalloc(&device->arguments.address_profile_1_1_524288, (argon2profile_1_1_524288.block_refs_size + 2) * 2 * sizeof(int32_t));
	if(device->error != hipSuccess) {
		device->error_message = "Error allocating memory.";
		return;
	}
	device->error = hipMemcpy(device->arguments.address_profile_1_1_524288, addresses_1_1_524288, argon2profile_1_1_524288.block_refs_size * 2 * sizeof(int32_t), hipMemcpyHostToDevice);
	if(device->error != hipSuccess) {
		device->error_message = "Error copying memory.";
		return;
	}
	free(addresses_1_1_524288);

	//optimise address sizes
	int16_t *addresses_4_4_16384 = (int16_t *)malloc(argon2profile_4_4_16384.block_refs_size * 2 * sizeof(int16_t));
	for(int i=0;i<argon2profile_4_4_16384.block_refs_size;i++) {
		addresses_4_4_16384[i*2] = argon2profile_4_4_16384.block_refs[i*3 + (i == 65528 ? 1 : 0)];
		addresses_4_4_16384[i*2 + 1] = argon2profile_4_4_16384.block_refs[i*3 + 2];
	}
	device->error = hipMalloc(&device->arguments.address_profile_4_4_16384, argon2profile_4_4_16384.block_refs_size * 2 * sizeof(int16_t));
	if(device->error != hipSuccess) {
		device->error_message = "Error allocating memory.";
		return;
	}
	device->error = hipMemcpy(device->arguments.address_profile_4_4_16384, addresses_4_4_16384, argon2profile_4_4_16384.block_refs_size * 2 * sizeof(int16_t), hipMemcpyHostToDevice);
	if(device->error != hipSuccess) {
		device->error_message = "Error copying memory.";
		return;
	}
	free(addresses_4_4_16384);

	//reorganize segments data
	uint16_t *segments_4_4_16384 = (uint16_t *)malloc(64 * 2 * sizeof(uint16_t));
	for(int i=0;i<64;i++) {
		int seg_start = argon2profile_4_4_16384.segments[i*3];
		segments_4_4_16384[i*2] = seg_start;
		segments_4_4_16384[i*2 + 1] = argon2profile_4_4_16384.block_refs[seg_start*3 + 1];
	}
	device->error = hipMalloc(&device->arguments.segments_profile_4_4_16384, 64 * 2 * sizeof(uint16_t));
	if(device->error != hipSuccess) {
		device->error_message = "Error allocating memory.";
		return;
	}
	device->error = hipMemcpy(device->arguments.segments_profile_4_4_16384, segments_4_4_16384, 64 * 2 * sizeof(uint16_t), hipMemcpyHostToDevice);
	if(device->error != hipSuccess) {
		device->error_message = "Error copying memory.";
		return;
	}
	free(segments_4_4_16384);

	device->error = hipMalloc(&device->arguments.offsets, 512 * sizeof(int));
	if(device->error != hipSuccess) {
		device->error_message = "Error allocating memory.";
		return;
	}
	device->error = hipMemcpy(device->arguments.offsets, offsets, 512 * sizeof(int), hipMemcpyHostToDevice);
	if(device->error != hipSuccess) {
		device->error_message = "Error copying memory.";
		return;
	}

	for(int i=0;i<device->device_threads;i++) {
		device->error = hipMalloc(&device->arguments.memory[i], device->threads_per_stream[i].memory_size);
		if(device->error != hipSuccess) {
			device->error_message = "Error allocating memory.";
			return;
		}

		size_t accessory_memory_size = max_threads * 8 * ARGON2_BLOCK_SIZE;
		device->error = hipMalloc(&device->arguments.seed_memory[i], accessory_memory_size);
		if (device->error != hipSuccess) {
			device->error_message = "Error allocating memory.";
			return;
		}
		device->error = hipMalloc(&device->arguments.out_memory[i], accessory_memory_size);
		if (device->error != hipSuccess) {
			device->error_message = "Error allocating memory.";
			return;
		}
		device->error = hipHostMalloc(&device->arguments.host_seed_memory[i], accessory_memory_size);
		if (device->error != hipSuccess) {
			device->error_message = "Error allocating pinned memory.";
			return;
		}
	}
}

void cuda_free(cuda_device_info *device) {
	hipSetDevice(device->device_index);

	if(device->arguments.address_profile_1_1_524288 != NULL) {
		hipFree(device->arguments.address_profile_1_1_524288);
		device->arguments.address_profile_1_1_524288 = NULL;
	}

	if(device->arguments.address_profile_4_4_16384 != NULL) {
		hipFree(device->arguments.address_profile_4_4_16384);
		device->arguments.address_profile_4_4_16384 = NULL;
	}

	if(device->arguments.segments_profile_4_4_16384 != NULL) {
		hipFree(device->arguments.segments_profile_4_4_16384);
		device->arguments.segments_profile_4_4_16384 = NULL;
	}

	if(device->arguments.offsets != NULL) {
		hipFree(device->arguments.offsets);
		device->arguments.offsets = NULL;
	}

	if(device->arguments.memory != NULL) {
		for(int i=0;i<device->device_threads;i++) {
			if(device->arguments.memory[i] != NULL)
				hipFree(device->arguments.memory[i]);
			device->arguments.memory[i] = NULL;
		}
	}

	if(device->arguments.seed_memory != NULL) {
		for(int i=0;i<device->device_threads;i++) {
			if(device->arguments.seed_memory[i] != NULL)
				hipFree(device->arguments.seed_memory[i]);
			device->arguments.seed_memory[i] = NULL;
		}
	}

	if(device->arguments.out_memory != NULL) {
		for(int i=0;i<device->device_threads;i++) {
			if(device->arguments.out_memory[i] != NULL)
				hipFree(device->arguments.out_memory[i]);
			device->arguments.out_memory[i] = NULL;
		}
	}

	if(device->arguments.host_seed_memory != NULL) {
		for(int i=0;i<device->device_threads;i++) {
			if(device->arguments.host_seed_memory[i] != NULL)
				hipHostFree(device->arguments.host_seed_memory[i]);
			device->arguments.host_seed_memory[i] = NULL;
		}
	}

	hipDeviceReset();
}

void *cuda_kernel_filler(void *memory, int threads, argon2profile *profile, void *user_data) {
	//    uint64_t start_log = microseconds();
	//    printf("Waiting for lock: %lld\n", microseconds() - start_log);
	//    start_log = microseconds();
	cuda_gpumgmt_thread_data *gpumgmt_thread = (cuda_gpumgmt_thread_data *)user_data;
	cuda_device_info *device = gpumgmt_thread->device;
	hipStream_t *stream = (hipStream_t *)gpumgmt_thread->cuda_info;

	int mem_seed_count = profile->thr_cost;
	size_t work_items;

	uint32_t memsize;
	uint32_t parallelism;

	if(strcmp(profile->profile_name, "1_1_524288") == 0) {
		memsize = (uint32_t)argon2profile_1_1_524288.memsize;
		parallelism = argon2profile_1_1_524288.thr_cost;
	}
	else {
		memsize = (uint32_t)argon2profile_4_4_16384.memsize;
		parallelism = argon2profile_4_4_16384.thr_cost;
	}
	work_items = KERNEL_WORKGROUP_SIZE * parallelism;

	device->error = hipMemcpyAsync(device->arguments.seed_memory[gpumgmt_thread->thread_id], memory, threads * 2 * mem_seed_count * ARGON2_BLOCK_SIZE, hipMemcpyHostToDevice, *stream);
	if (device->error != hipSuccess) {
		device->error_message = "Error writing to gpu memory.";
		return NULL;
	}

	if(parallelism == 1) {
		fill_blocks_cpu<<<threads, work_items, 0, *stream>>>((uint64_t*)device->arguments.memory[gpumgmt_thread->thread_id],
				device->arguments.seed_memory[gpumgmt_thread->thread_id],
				device->arguments.out_memory[gpumgmt_thread->thread_id],
				device->arguments.address_profile_1_1_524288,
				device->arguments.offsets,
				memsize);
	}
	else {
		fill_blocks_gpu<< < threads, work_items, 0, *stream >> > ((uint64_t *) device->arguments.memory[gpumgmt_thread->thread_id],
				device->arguments.seed_memory[gpumgmt_thread->thread_id],
				device->arguments.out_memory[gpumgmt_thread->thread_id],
				device->arguments.address_profile_4_4_16384,
				device->arguments.segments_profile_4_4_16384,
				device->arguments.offsets,
				memsize);
	}

	device->error = hipMemcpyAsync(memory, device->arguments.out_memory[gpumgmt_thread->thread_id], threads * 2 * mem_seed_count * ARGON2_BLOCK_SIZE, hipMemcpyDeviceToHost, *stream);
	if (device->error != hipSuccess) {
		device->error_message = "Error reading gpu memory.";
		return NULL;
	}

	while(hipStreamQuery(*stream) != hipSuccess) {
		this_thread::sleep_for(chrono::milliseconds(10));
		continue;
	}

	return memory;
}