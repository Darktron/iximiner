#include "hip/hip_runtime.h"
#include <hip/driver_types.h>
#include "../../../common/common.h"
#include "../../../app/arguments.h"

#include "../../hasher.h"
#include "../../argon2/argon2.h"

#include "cuda_hasher.h"

#define MEMORY_CHUNK_PER_ITEM           4
#define ITEMS_PER_SEGMENT               32
#define BLOCK_SIZE_ULONG                128
#define KERNEL_WORKGROUP_SIZE   		32

__device__ uint64_t upsample(uint32_t hi, uint32_t lo)
{
	return ((uint64_t)hi << 32) | (uint64_t)lo;
}

__device__ uint64_t rotate(uint64_t x, uint32_t n)
{
	return (x >> (64-n)) | (x << n);
}

#define fBlaMka(x, y) ((x) + (y) + 2 * upsample(__umulhi((uint32_t)(x), (uint32_t)(y)), (uint32_t)(x) * (uint32_t)y))

#if defined(USE_NON_PTX_CUDA)

	#define COMPUTE            \
		a = fBlaMka(a, b);          \
		d = rotate(d ^ a, 32);      \
		c = fBlaMka(c, d);          \
		b = rotate(b ^ c, 40);      \
		a = fBlaMka(a, b);          \
		d = rotate(d ^ a, 48);      \
		c = fBlaMka(c, d);          \
		b = rotate(b ^ c, 1);
#else

    #define COMPUTE            \
	asm ("{"  \
	     ".reg .u64 d1, d2, a, b, c, d;\n\t"     \
		 ".reg .u32 s1, s2, s3, s4;\n\t"     \
		 "add.u64 d1, %0, %1;\n\t"     \
		 "cvt.u32.u64 s1, %0;\n\t"     \
		 "cvt.u32.u64 s2, %1;\n\t"     \
		 "mul.lo.u32 s3, s1, s2;\n\t"     \
		 "mul.hi.u32 s4, s1, s2;\n\t"     \
		 "mov.b64 a, {s3, s4};\n\t"     \
		 "shl.b64 d2, a, 1;\n\t"     \
		 "add.u64 a, d1, d2;\n\t"     \
		 "xor.b64 d1, %3, a;\n\t"     \
		 "mov.b64 {s1, s2}, d1;\n\t"     \
		 "mov.b64 d, {s2, s1};\n\t"     \
		 "add.u64 d1, %2, d;\n\t"     \
		 "cvt.u32.u64 s1, %2;\n\t"     \
		 "mul.lo.u32 s3, s1, s2;\n\t"     \
		 "mul.hi.u32 s4, s1, s2;\n\t"     \
		 "mov.b64 c, {s3, s4};\n\t"     \
		 "shl.b64 d2, c, 1;\n\t"     \
		 "add.u64 c, d1, d2;\n\t"     \
		 "xor.b64 d1, %1, c;\n\t"     \
		 "mov.b64 {s3, s4}, d1;\n\t"     \
		 "prmt.b32 s2, s3, s4, 0x6543;\n\t"     \
		 "prmt.b32 s1, s3, s4, 0x2107;\n\t"     \
		 "mov.b64 b, {s2, s1};\n\t"     \
		 "add.u64 d1, a, b;\n\t"     \
		 "cvt.u32.u64 s1, a;\n\t"     \
		 "mul.lo.u32 s3, s1, s2;\n\t"     \
		 "mul.hi.u32 s4, s1, s2;\n\t"     \
		 "mov.b64 a, {s3, s4};\n\t"     \
		 "shl.b64 d2, a, 1;\n\t"     \
		 "add.u64 %0, d1, d2;\n\t"     \
		 "xor.b64 d1, d, %0;\n\t"     \
		 "mov.b64 {s3, s4}, d1;\n\t"     \
		 "prmt.b32 s2, s3, s4, 0x5432;\n\t"     \
		 "prmt.b32 s1, s3, s4, 0x1076;\n\t"     \
		 "mov.b64 %3, {s2, s1};\n\t"     \
		 "add.u64 d1, c, %3;\n\t"     \
		 "cvt.u32.u64 s1, c;\n\t"     \
		 "mul.lo.u32 s3, s1, s2;\n\t"     \
		 "mul.hi.u32 s4, s1, s2;\n\t"     \
		 "mov.b64 c, {s3, s4};\n\t"     \
		 "shl.b64 d2, c, 1;\n\t"     \
		 "add.u64 %2, d1, d2;\n\t"     \
		 "xor.b64 d1, b, %2;\n\t"     \
		 "shl.b64 a, d1, 1;\n\t"     \
		 "shr.b64 b, d1, 63;\n\t"     \
		 "add.u64 %1, a, b;\n\t" \
	"}" : "+l"(a), "+l"(b), "+l"(c), "+l"(d));

#endif

#define G1(data)           \
{                           \
	COMPUTE \
	data[i1_1] = b; \
    data[i1_2] = c; \
    data[i1_3] = d; \
    __syncthreads(); \
}

#define G2(data)           \
{ \
    b = data[i2_1]; \
    c = data[i2_2]; \
    d = data[i2_3]; \
	COMPUTE \
    data[i2_0] = a; \
    data[i2_1] = b; \
    data[i2_2] = c; \
    data[i2_3] = d; \
    __syncthreads(); \
}

#define G3(data)           \
{                           \
    a = data[i3_0]; \
    b = data[i3_1]; \
    c = data[i3_2]; \
    d = data[i3_3]; \
	COMPUTE \
	data[i3_1] = b; \
    data[i3_2] = c; \
    data[i3_3] = d; \
    __syncthreads(); \
}

#define G4(data)           \
{                           \
    b = data[i4_1]; \
    c = data[i4_2]; \
    d = data[i4_3]; \
	COMPUTE \
    data[i4_0] = a; \
    data[i4_1] = b; \
    data[i4_2] = c; \
    data[i4_3] = d; \
    __syncthreads(); \
    a = data[i1_0]; \
    b = data[i1_1]; \
    c = data[i1_2]; \
    d = data[i1_3]; \
}

#define copy_block(dst, src) for(int i=0;i<4;i++) (dst)[i] = (src)[i]
#define xor_block(dst, src) for(int i=0;i<4;i++) (dst)[i] ^= (src)[i]

#define copy_block_small(dst, src) dst[local_id] = src[local_id]
#define xor_block_small(dst, src) dst[local_id] ^= src[local_id]

int offsets[512] = {
		0, 4, 8, 12,
		1, 5, 9, 13,
		2, 6, 10, 14,
		3, 7, 11, 15,
		16, 20, 24, 28,
		17, 21, 25, 29,
		18, 22, 26, 30,
		19, 23, 27, 31,
		32, 36, 40, 44,
		33, 37, 41, 45,
		34, 38, 42, 46,
		35, 39, 43, 47,
		48, 52, 56, 60,
		49, 53, 57, 61,
		50, 54, 58, 62,
		51, 55, 59, 63,
		64, 68, 72, 76,
		65, 69, 73, 77,
		66, 70, 74, 78,
		67, 71, 75, 79,
		80, 84, 88, 92,
		81, 85, 89, 93,
		82, 86, 90, 94,
		83, 87, 91, 95,
		96, 100, 104, 108,
		97, 101, 105, 109,
		98, 102, 106, 110,
		99, 103, 107, 111,
		112, 116, 120, 124,
		113, 117, 121, 125,
		114, 118, 122, 126,
		115, 119, 123, 127,
		0, 5, 10, 15,
		1, 6, 11, 12,
		2, 7, 8, 13,
		3, 4, 9, 14,
		16, 21, 26, 31,
		17, 22, 27, 28,
		18, 23, 24, 29,
		19, 20, 25, 30,
		32, 37, 42, 47,
		33, 38, 43, 44,
		34, 39, 40, 45,
		35, 36, 41, 46,
		48, 53, 58, 63,
		49, 54, 59, 60,
		50, 55, 56, 61,
		51, 52, 57, 62,
		64, 69, 74, 79,
		65, 70, 75, 76,
		66, 71, 72, 77,
		67, 68, 73, 78,
		80, 85, 90, 95,
		81, 86, 91, 92,
		82, 87, 88, 93,
		83, 84, 89, 94,
		96, 101, 106, 111,
		97, 102, 107, 108,
		98, 103, 104, 109,
		99, 100, 105, 110,
		112, 117, 122, 127,
		113, 118, 123, 124,
		114, 119, 120, 125,
		115, 116, 121, 126,
		0, 32, 64, 96,
		1, 33, 65, 97,
		2, 34, 66, 98,
		3, 35, 67, 99,
		4, 36, 68, 100,
		5, 37, 69, 101,
		6, 38, 70, 102,
		7, 39, 71, 103,
		8, 40, 72, 104,
		9, 41, 73, 105,
		10, 42, 74, 106,
		11, 43, 75, 107,
		12, 44, 76, 108,
		13, 45, 77, 109,
		14, 46, 78, 110,
		15, 47, 79, 111,
		16, 48, 80, 112,
		17, 49, 81, 113,
		18, 50, 82, 114,
		19, 51, 83, 115,
		20, 52, 84, 116,
		21, 53, 85, 117,
		22, 54, 86, 118,
		23, 55, 87, 119,
		24, 56, 88, 120,
		25, 57, 89, 121,
		26, 58, 90, 122,
		27, 59, 91, 123,
		28, 60, 92, 124,
		29, 61, 93, 125,
		30, 62, 94, 126,
		31, 63, 95, 127,
		0, 33, 80, 113,
		1, 48, 81, 96,
		2, 35, 82, 115,
		3, 50, 83, 98,
		4, 37, 84, 117,
		5, 52, 85, 100,
		6, 39, 86, 119,
		7, 54, 87, 102,
		8, 41, 88, 121,
		9, 56, 89, 104,
		10, 43, 90, 123,
		11, 58, 91, 106,
		12, 45, 92, 125,
		13, 60, 93, 108,
		14, 47, 94, 127,
		15, 62, 95, 110,
		16, 49, 64, 97,
		17, 32, 65, 112,
		18, 51, 66, 99,
		19, 34, 67, 114,
		20, 53, 68, 101,
		21, 36, 69, 116,
		22, 55, 70, 103,
		23, 38, 71, 118,
		24, 57, 72, 105,
		25, 40, 73, 120,
		26, 59, 74, 107,
		27, 42, 75, 122,
		28, 61, 76, 109,
		29, 44, 77, 124,
		30, 63, 78, 111,
		31, 46, 79, 126
};

__global__ void fill_blocks_cpu(uint64_t *scratchpad,
							uint64_t *seed,
							uint64_t *out,
							int *addresses,
							int *offsets_,
							int memsize) {
	__shared__ uint64_t state[BLOCK_SIZE_ULONG];
	uint64_t a, b, c, d, x, y, z, w;

	int hash = blockIdx.x;
	int local_id = threadIdx.x;

	int offset = local_id << 2;

	uint64_t *memory = scratchpad + hash * (memsize >> 3);

	uint64_t *out_mem = out + hash * 2 * BLOCK_SIZE_ULONG;
	uint64_t *mem_seed = seed + hash * 2 * BLOCK_SIZE_ULONG;

	uint64_t *seed_dst = memory;
	copy_block(&seed_dst[offset], &mem_seed[offset]);
	mem_seed += BLOCK_SIZE_ULONG;
	seed_dst += BLOCK_SIZE_ULONG;
	copy_block(&seed_dst[offset], &mem_seed[offset]);

	uint64_t *next_block;
	uint64_t *ref_block;

	int id_4 = local_id << 2;
	int i1_0 = offsets_[id_4];
	int i1_1 = offsets_[id_4 + 1];
	int i1_2 = offsets_[id_4 + 2];
	int i1_3 = offsets_[id_4 + 3];

	int i2_0 = offsets_[id_4 + 128];
	int i2_1 = offsets_[id_4 + 129];
	int i2_2 = offsets_[id_4 + 130];
	int i2_3 = offsets_[id_4 + 131];

	int i3_0 = offsets_[id_4 + 256];
	int i3_1 = offsets_[id_4 + 257];
	int i3_2 = offsets_[id_4 + 258];
	int i3_3 = offsets_[id_4 + 259];

	int i4_0 = offsets_[id_4 + 384];
	int i4_1 = offsets_[id_4 + 385];
	int i4_2 = offsets_[id_4 + 386];
	int i4_3 = offsets_[id_4 + 387];

	int *stop_addr = addresses + 1572858;

	a = seed_dst[i1_0];
	b = seed_dst[i1_1];
	c = seed_dst[i1_2];
	d = seed_dst[i1_3];

	for(; addresses < stop_addr; addresses += 3) {
		__syncthreads();

		int addr0 = addresses[0];
		int addr2 = addresses[2];

		if(addr0 != -1) {
			next_block = memory + addr0 * BLOCK_SIZE_ULONG;
		}
		ref_block = memory + addr2 * BLOCK_SIZE_ULONG;

		x = a = a ^ ref_block[i1_0];
		y = b = b ^ ref_block[i1_1];
		z = c = c ^ ref_block[i1_2];
		w = d = d ^ ref_block[i1_3];

		G1(state);
		G2(state);
		G3(state);
		G4(state);

		a ^= x;
		b ^= y;
		c ^= z;
		d ^= w;

		if(addr0 != -1) {
			next_block[i1_0] = a;
			next_block[i1_1] = b;
			next_block[i1_2] = c;
			next_block[i1_3] = d;
		}
	}

	out_mem[i1_0] = a;
	out_mem[i1_1] = b;
	out_mem[i1_2] = c;
	out_mem[i1_3] = d;
};

__global__ void fill_blocks(uint64_t *scratchpad,
							uint64_t *out,
							int *refs_1_1_524288,
							int *refs_4_4_16384,
							int *seg_1_1_524288,
							int *seg_4_4_16384,
							int *offsets_,
							uint64_t *seed,
							int memsize,
							int addrsize,
							int parallelism) {
	__shared__ uint64_t state[4 * BLOCK_SIZE_ULONG];
	uint64_t a, b, c, d, x, y, z, w;

	int hash = blockIdx.x;
	int local_id = threadIdx.x;

	int id = local_id % ITEMS_PER_SEGMENT;
	int segment = local_id / ITEMS_PER_SEGMENT;
	int offset = id * MEMORY_CHUNK_PER_ITEM;

	int *addresses = parallelism == 1 ? refs_1_1_524288 : refs_4_4_16384;
	int *segments = parallelism == 1 ? seg_1_1_524288 : seg_4_4_16384;
	int segments_in_lane = parallelism == 1 ? 1 : 16;

	uint64_t *memory = scratchpad + hash * (memsize >> 3);

	int lane_length = parallelism == 1 ? 0 : 4096;

	uint64_t *out_mem = out + hash * 2 * parallelism * BLOCK_SIZE_ULONG;
	uint64_t *mem_seed = seed + hash * 2 * parallelism * BLOCK_SIZE_ULONG;

	uint64_t *seed_src = mem_seed + segment * 2 * BLOCK_SIZE_ULONG;
	uint64_t *seed_dst = memory + segment * lane_length * BLOCK_SIZE_ULONG;
	copy_block(&seed_dst[offset], &seed_src[offset]);
	seed_src += BLOCK_SIZE_ULONG;
	seed_dst += BLOCK_SIZE_ULONG;
	copy_block(&seed_dst[offset], &seed_src[offset]);

	uint64_t *next_block;
	uint64_t *prev_block;
	uint64_t *ref_block;

	uint64_t *local_state = state + segment * BLOCK_SIZE_ULONG;

	int id_4 = id << 2;
	int i1_0 = offsets_[id_4];
	int i1_1 = offsets_[id_4 + 1];
	int i1_2 = offsets_[id_4 + 2];
	int i1_3 = offsets_[id_4 + 3];

	int i2_0 = offsets_[id_4 + 128];
	int i2_1 = offsets_[id_4 + 129];
	int i2_2 = offsets_[id_4 + 130];
	int i2_3 = offsets_[id_4 + 131];

	int i3_0 = offsets_[id_4 + 256];
	int i3_1 = offsets_[id_4 + 257];
	int i3_2 = offsets_[id_4 + 258];
	int i3_3 = offsets_[id_4 + 259];

	int i4_0 = offsets_[id_4 + 384];
	int i4_1 = offsets_[id_4 + 385];
	int i4_2 = offsets_[id_4 + 386];
	int i4_3 = offsets_[id_4 + 387];

	for(int s=0; s<segments_in_lane; s++) {
		int *curr_seg = segments  + 3 * (s * parallelism + segment);
		int *addr = addresses + 3 * curr_seg[0];
		int *stop_addr = addresses + 3 * curr_seg[1];
		int with_xor = curr_seg[2];

		for(; addr < stop_addr; addr += 3) {
			__syncthreads();

			int addr0 = addr[0];
			int addr1 = addr[1];
			int addr2 = addr[2];

			if(addr0 != -1) {
				next_block = memory + addr0 * BLOCK_SIZE_ULONG;
			}
			if(addr1 != -1) {
				prev_block = memory + addr1 * BLOCK_SIZE_ULONG;
				a = prev_block[i1_0];
				b = prev_block[i1_1];
				c = prev_block[i1_2];
				d = prev_block[i1_3];
			}
			ref_block = memory + addr2 * BLOCK_SIZE_ULONG;

			x = a = a ^ ref_block[i1_0];
			y = b = b ^ ref_block[i1_1];
			z = c = c ^ ref_block[i1_2];
			w = d = d ^ ref_block[i1_3];

			if(with_xor == 1) {
				x ^= next_block[i1_0];
				y ^= next_block[i1_1];
				z ^= next_block[i1_2];
				w ^= next_block[i1_3];
			}

			G1(local_state);
			G2(local_state);
			G3(local_state);
			G4(local_state);

			a ^= x;
			b ^= y;
			c ^= z;
			d ^= w;

			if(addr0 != -1) {
				next_block[i1_0] = a;
				next_block[i1_1] = b;
				next_block[i1_2] = c;
				next_block[i1_3] = d;
			}
		}
	}

	__syncthreads();
	int dst_addr = (parallelism == 1) ? addrsize : (addrsize - 3);

	int result_block = (parallelism == 1) ? 0 : addresses[dst_addr * 3 + 1];

	next_block = memory + result_block * BLOCK_SIZE_ULONG;
	if(parallelism == 1) {
		copy_block(&out_mem[offset], &next_block[offset]);
	}
	else {
		copy_block_small(out_mem, next_block);
	}

	for(;dst_addr < addrsize; ++dst_addr) {
		next_block = memory + addresses[dst_addr * 3 + 2] * BLOCK_SIZE_ULONG;
		if(parallelism == 1) {
			xor_block(&out_mem[offset], &next_block[offset]);
		}
		else {
			xor_block_small(out_mem, next_block);
		}
	}
};

void cuda_allocate(cuda_device_info *device) {
	int max_threads = max(device->threads_profile_1_1_524288, device->threads_profile_4_4_16384);

	device->error = hipSetDevice(device->device_index);
	if(device->error != hipSuccess) {
		device->error_message = "Error setting current device for memory allocation.";
		return;
	}

	device->error = hipMalloc(&device->arguments.memory, device->arguments.memory_size);
	if(device->error != hipSuccess) {
		device->error_message = "Error allocating memory.";
		return;
	}

	device->error = hipMalloc(&device->arguments.address_profile_1_1_524288, argon2profile_1_1_524288.block_refs_size * 3 * sizeof(int32_t));
	if(device->error != hipSuccess) {
		device->error_message = "Error allocating memory.";
		return;
	}
	device->error = hipMemcpy(device->arguments.address_profile_1_1_524288, argon2profile_1_1_524288.block_refs, argon2profile_1_1_524288.block_refs_size * 3 * sizeof(int32_t), hipMemcpyHostToDevice);
	if(device->error != hipSuccess) {
		device->error_message = "Error copying memory.";
		return;
	}

	device->error = hipMalloc(&device->arguments.address_profile_4_4_16384, argon2profile_4_4_16384.block_refs_size * 3 * sizeof(int32_t));
	if(device->error != hipSuccess) {
		device->error_message = "Error allocating memory.";
		return;
	}
	device->error = hipMemcpy(device->arguments.address_profile_4_4_16384, argon2profile_4_4_16384.block_refs, argon2profile_4_4_16384.block_refs_size * 3 * sizeof(int32_t), hipMemcpyHostToDevice);
	if(device->error != hipSuccess) {
		device->error_message = "Error copying memory.";
		return;
	}

	device->error = hipMalloc(&device->arguments.segments_profile_1_1_524288, 3 * sizeof(int32_t));
	if(device->error != hipSuccess) {
		device->error_message = "Error allocating memory.";
		return;
	}
	device->error = hipMemcpy(device->arguments.segments_profile_1_1_524288, argon2profile_1_1_524288.segments, 3 * sizeof(int32_t), hipMemcpyHostToDevice);
	if(device->error != hipSuccess) {
		device->error_message = "Error copying memory.";
		return;
	}

	device->error = hipMalloc(&device->arguments.segments_profile_4_4_16384, 64 * 3 * sizeof(int32_t));
	if(device->error != hipSuccess) {
		device->error_message = "Error allocating memory.";
		return;
	}
	device->error = hipMemcpy(device->arguments.segments_profile_4_4_16384, argon2profile_4_4_16384.segments, 64 * 3 * sizeof(int32_t), hipMemcpyHostToDevice);
	if(device->error != hipSuccess) {
		device->error_message = "Error copying memory.";
		return;
	}

	device->error = hipMalloc(&device->arguments.offsets, 512 * sizeof(int));
	if(device->error != hipSuccess) {
		device->error_message = "Error allocating memory.";
		return;
	}
	device->error = hipMemcpy(device->arguments.offsets, offsets, 512 * sizeof(int), hipMemcpyHostToDevice);
	if(device->error != hipSuccess) {
		device->error_message = "Error copying memory.";
		return;
	}

	device->error = hipMalloc(&device->arguments.seed_memory[0], max_threads * 8 * ARGON2_BLOCK_SIZE);
	if(device->error != hipSuccess) {
		device->error_message = "Error allocating memory.";
		return;
	}
	device->error = hipMalloc(&device->arguments.seed_memory[1], max_threads * 8 * ARGON2_BLOCK_SIZE);
	if(device->error != hipSuccess) {
		device->error_message = "Error allocating memory.";
		return;
	}
	device->error = hipMalloc(&device->arguments.out_memory[0], max_threads * 8 * ARGON2_BLOCK_SIZE);
	if(device->error != hipSuccess) {
		device->error_message = "Error allocating memory.";
		return;
	}
	device->error = hipMalloc(&device->arguments.out_memory[1], max_threads * 8 * ARGON2_BLOCK_SIZE);
	if(device->error != hipSuccess) {
		device->error_message = "Error allocating memory.";
		return;
	}

	device->error = hipHostMalloc(&device->arguments.host_seed_memory[0], max_threads * 8 * ARGON2_BLOCK_SIZE);
	if(device->error != hipSuccess) {
		device->error_message = "Error allocating pinned memory.";
		return;
	}
	device->error = hipHostMalloc(&device->arguments.host_seed_memory[1], max_threads * 8 * ARGON2_BLOCK_SIZE);
	if(device->error != hipSuccess) {
		device->error_message = "Error allocating pinned memory.";
		return;
	}
}

void cuda_free(cuda_device_info *device) {
	hipSetDevice(device->device_index);

	if(device->arguments.memory != NULL)
		hipFree(device->arguments.memory);
	if(device->arguments.address_profile_1_1_524288 != NULL)
		hipFree(device->arguments.address_profile_1_1_524288);
	if(device->arguments.address_profile_4_4_16384 != NULL)
		hipFree(device->arguments.address_profile_4_4_16384);
	if(device->arguments.segments_profile_1_1_524288 != NULL)
		hipFree(device->arguments.segments_profile_1_1_524288);
	if(device->arguments.segments_profile_4_4_16384 != NULL)
		hipFree(device->arguments.segments_profile_4_4_16384);
	if(device->arguments.seed_memory[0] != NULL)
		hipFree(device->arguments.seed_memory[0]);
	if(device->arguments.seed_memory[1] != NULL)
		hipFree(device->arguments.seed_memory[1]);
	if(device->arguments.out_memory[0] != NULL)
		hipFree(device->arguments.out_memory[0]);
	if(device->arguments.out_memory[1] != NULL)
		hipFree(device->arguments.out_memory[1]);
	if(device->arguments.offsets != NULL)
		hipFree(device->arguments.offsets);
	if(device->arguments.host_seed_memory[0] != NULL)
		hipHostFree(device->arguments.host_seed_memory[0]);
	if(device->arguments.host_seed_memory[1] != NULL)
		hipHostFree(device->arguments.host_seed_memory[1]);

}

void *cuda_kernel_filler(void *memory, int threads, argon2profile *profile, void *user_data) {
	//    uint64_t start_log = microseconds();
	//    printf("Waiting for lock: %lld\n", microseconds() - start_log);
	//    start_log = microseconds();
	cuda_gpumgmt_thread_data *gpumgmt_thread = (cuda_gpumgmt_thread_data *)user_data;
	cuda_device_info *device = gpumgmt_thread->device;

	int mem_seed_count = profile->thr_cost;
	size_t work_items;

	uint32_t memsize;
	uint32_t addrsize;
	uint32_t parallelism;

	if(strcmp(profile->profile_name, "1_1_524288") == 0) {
		memsize = (uint32_t)argon2profile_1_1_524288.memsize;
		addrsize = (uint32_t)argon2profile_1_1_524288.block_refs_size;
		parallelism = argon2profile_1_1_524288.thr_cost;
	}
	else {
		memsize = (uint32_t)argon2profile_4_4_16384.memsize;
		addrsize = (uint32_t)argon2profile_4_4_16384.block_refs_size;
		parallelism = argon2profile_4_4_16384.thr_cost;
	}
	work_items = KERNEL_WORKGROUP_SIZE * parallelism;

	device->device_lock.lock();

	device->error = hipMemcpy(device->arguments.seed_memory[gpumgmt_thread->thread_id], memory, threads * 2 * mem_seed_count * ARGON2_BLOCK_SIZE, hipMemcpyHostToDevice);
	if (device->error != hipSuccess) {
		device->error_message = "Error writing to gpu memory.";
		device->device_lock.unlock();
		return NULL;
	}

	if(parallelism == 1) {
		fill_blocks_cpu<<<threads, work_items>>>((uint64_t*)device->arguments.memory,
				device->arguments.seed_memory[gpumgmt_thread->thread_id],
				device->arguments.out_memory[gpumgmt_thread->thread_id],
				device->arguments.address_profile_1_1_524288,
				device->arguments.offsets,
				memsize);
	}
	else {
		fill_blocks << < threads, work_items >> > ((uint64_t *) device->arguments.memory,
				device->arguments.out_memory[gpumgmt_thread->thread_id],
				device->arguments.address_profile_1_1_524288,
				device->arguments.address_profile_4_4_16384,
				device->arguments.segments_profile_1_1_524288,
				device->arguments.segments_profile_4_4_16384,
				device->arguments.offsets,
				device->arguments.seed_memory[gpumgmt_thread->thread_id],
				memsize,
				addrsize,
				parallelism);
	}

	device->error = hipMemcpy(memory, device->arguments.out_memory[gpumgmt_thread->thread_id], threads * 2 * mem_seed_count * ARGON2_BLOCK_SIZE, hipMemcpyDeviceToHost);
	if (device->error != hipSuccess) {
		device->error_message = "Error reading gpu memory.";
		device->device_lock.unlock();
		return NULL;
	}

	device->device_lock.unlock();

	return memory;
}